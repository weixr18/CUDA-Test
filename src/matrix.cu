#include "hip/hip_runtime.h"
#include <stdio.h>
#include <Windows.h>
#include "matrix.h"

#define __TEST__

#ifdef __cplusplus
extern "C"{
#endif


// Forward declaration of the matrix multiplication kernel    
__global__ void MatMulKernel(const Matrix, const Matrix, Matrix); 
    
// Matrix multiplication - Host code    
// Matrix dimensions are assumed to be multiples of BLOCK_SIZE    
void MatMul(const Matrix A, const Matrix B, Matrix C) {   
    
    // dimension check
    if(A.width != B.height){
        printf("A dimension 1 doesn't match B dimension 0.\n");
        return;
    }
    if(A.height != C.height){
        printf("A dimension 0 doesn't match C dimension 0.\n");
        return;
    }
    if(B.width != C.width){
        printf("B dimension 1 doesn't match C dimension 1.\n");
        return;
    }

    // Load A and B to device memory    
    Matrix d_A;    
    d_A.width = A.width; d_A.height = A.height;    
    size_t size = A.width * A.height * sizeof(float);    
    hipMalloc(&d_A.elements, size);    
    hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);    
    Matrix d_B;    
    d_B.width = B.width; d_B.height = B.height;    
    size = B.width * B.height * sizeof(float);    
    hipMalloc(&d_B.elements, size);    
    hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);    
    
    // Allocate C in device memory    
    Matrix d_C;    
    d_C.width = C.width; d_C.height = C.height;    
    size = C.width * C.height * sizeof(float);    
    hipMalloc(&d_C.elements, size);    
    
    // Invoke kernel    
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);    
    dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);   
    
#ifdef __TEST__
    // time test code
    LARGE_INTEGER nFreq;
    LARGE_INTEGER t1;
    LARGE_INTEGER t2;
    double dt;
    QueryPerformanceFrequency(&nFreq);
    QueryPerformanceCounter(&t1);
#endif //__TEST

    MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);  

#ifdef __TEST__
    QueryPerformanceCounter(&t2);
    dt = (t2.QuadPart -t1.QuadPart)/(double)nFreq.QuadPart; 
    printf("%lfs\n", dt);  
#endif //__TEST
    
    // Read C from device memory    
    hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);    
    
    // Free device memory    
    hipFree(d_A.elements);    
    hipFree(d_B.elements);    
    hipFree(d_C.elements);    
}    
    
// Matrix multiplication kernel called by MatMul()    
__global__ void MatMulKernel(Matrix A, Matrix B, Matrix C) {    
    // Each thread computes one element of C    
    // by accumulating results into Cvalue    
    float Cvalue = 0;    
    int row  = blockIdx.y * blockDim.y + threadIdx.y;    
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    for (int e = 0; e < A.width; ++e) {   
		Cvalue += A.elements[row * A.width + e] * B.elements[e * B.width + col];    
	}
    C.elements[row * C.width + col] = Cvalue;    
}

#ifdef __cplusplus
};
#endif